#include "hip/hip_runtime.h"
#include <fstream>
#include "camera_t.h"
#include "scene_t.h"
#include "render.h"

__global__ void render_kernel(camera_t* d_camera, scene_t* d_scene, vec3_t* d_framebuffer,
                              unsigned int image_width, unsigned int image_height) {
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int thread_id = y * gridDim.x * blockDim.x + x;
    if (x >= image_width || y >= image_height)
        return;

    hiprandState rand_state;
    hiprand_init(0, thread_id, 0, &rand_state);
    float s = float(x) / float(image_width - 1);
    float t = 1.0f - float(y) / float(image_height - 1);
    ray_t camera_ray = d_camera->get_ray(s, t);
    vec3_t color = vec3_t::make_zeros();
    for (int k = 1; k <= SAMPLES_PER_PIXEL; k++)
        color = color + get_color(*d_scene, camera_ray, rand_state);
    d_framebuffer[y * image_width + x] = color / SAMPLES_PER_PIXEL;
}

int main() {
    // camera
    vec3_t lookfrom(0.5f, 0.5f, 1.0f);
    vec3_t lookat(0.5f, 0.5f, 0.0f);
    vec3_t vup(0.0f, 1.0f, 0.0f);
    float vfov = 55.0f;
    float aspect_ratio = 1.0f;
    camera_t camera(lookfrom, lookat, vup, vfov, aspect_ratio);
    camera_t* d_camera;
    CHECK_CUDA(hipMalloc(&d_camera, sizeof(camera_t)));
    CHECK_CUDA(hipMemcpy(d_camera, &camera, sizeof(camera_t), hipMemcpyHostToDevice));

    // color
    vec3_t red(0.65f, 0.05f, 0.05f);
    vec3_t green(0.12f, 0.45f, 0.15f);
    vec3_t white(0.73f, 0.73f, 0.73f);
    vec3_t brown(0.62f, 0.57f, 0.54f);

    // scene
    std::vector<sphere_t> spheres;
    std::vector<trig_t> trigs;
    auto add_rectangle = [&](const vec3_t &p0, const vec3_t &p1, const vec3_t &p2,
                             const vec3_t &p3, const vec3_t &albedo) {
        trigs.emplace_back(p0, p1, p2, albedo);
        trigs.emplace_back(p2, p3, p0, albedo);
    };
    spheres.emplace_back(vec3_t(0.5f, 0.2f, -0.25f), 0.2f, brown);
    add_rectangle(vec3_t(0.0f, 0.0f, 0.0f),
                  vec3_t(0.0f, 1.0f, 0.0f),
                  vec3_t(0.0f, 1.0f, -1.0f),
                  vec3_t(0.0f, 0.0f, -1.0f),
                  red);
    add_rectangle(vec3_t(0.0f, 0.0f, -1.0f),
                  vec3_t(0.0f, 1.0f, -1.0f),
                  vec3_t(1.0f, 1.0f, -1.0f),
                  vec3_t(1.0f, 0.0f, -1.0f),
                  white);
    add_rectangle(vec3_t(1.0f, 0.0f, 0.0f),
                  vec3_t(1.0f, 1.0f, 0.0f),
                  vec3_t(1.0f, 1.0f, -1.0f),
                  vec3_t(1.0f, 0.0f, -1.0f),
                  green);
    add_rectangle(vec3_t(0.0f, 1.0f, 0.0f),
                  vec3_t(0.0f, 1.0f, -1.0f),
                  vec3_t(1.0f, 1.0f, -1.0f),
                  vec3_t(1.0f, 1.0f, 0.0f),
                  white);
    add_rectangle(vec3_t(0.0f, 0.0f, 0.0f),
                  vec3_t(0.0f, 0.0f, -1.0f),
                  vec3_t(1.0f, 0.0f, -1.0f),
                  vec3_t(1.0f, 0.0f, 0.0f),
                  white);
    scene_t scene{};
    scene.num_spheres = (int)spheres.size();
    CHECK_CUDA(hipMalloc(&scene.spheres, scene.num_spheres * sizeof(sphere_t)));
    CHECK_CUDA(hipMemcpy(scene.spheres, spheres.data(), scene.num_spheres * sizeof(sphere_t), hipMemcpyHostToDevice));
    scene.num_trigs = (int)trigs.size();
    CHECK_CUDA(hipMalloc(&scene.trigs, scene.num_trigs * sizeof(trig_t)));
    CHECK_CUDA(hipMemcpy(scene.trigs, trigs.data(), scene.num_trigs * sizeof(trig_t), hipMemcpyHostToDevice));
    scene.point_light = {vec3_t(0.95f, 0.95f, 0.3f), vec3_t(0.9f, 0.9f, 0.9f)};
    scene_t* d_scene;
    CHECK_CUDA(hipMalloc(&d_scene, sizeof(scene_t)));
    CHECK_CUDA(hipMemcpy(d_scene, &scene, sizeof(scene_t), hipMemcpyHostToDevice));

    // render
    unsigned int image_width = 600;
    unsigned int image_height = 600;

    vec3_t* d_framebuffer;
    CHECK_CUDA(hipMalloc(&d_framebuffer, image_height * image_width * sizeof(vec3_t)));
    dim3 block_size(BLOCK_SIZE_X, BLOCK_SIZE_Y);
    dim3 grid_size((image_width + block_size.x - 1) / block_size.x, (image_height + block_size.y - 1) / block_size.y);
    render_kernel<<<grid_size, block_size>>>(d_camera, d_scene, d_framebuffer, image_width, image_height);

    // write framebuffer to file
    vec3_t framebuffer[image_height * image_width];
    CHECK_CUDA(hipMemcpy(framebuffer, d_framebuffer, image_height * image_width * sizeof(vec3_t), hipMemcpyDeviceToHost));
    std::ofstream image_fs("image.ppm");
    image_fs << "P3\n" << image_width << ' ' << image_height << "\n255\n";
    for (int i = 0; i < image_height; i++)
        for (int j = 0; j < image_width; j++)
            framebuffer[i * image_width + j].write_color(image_fs);

    return 0;
}
