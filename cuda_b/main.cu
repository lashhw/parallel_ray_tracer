#include "hip/hip_runtime.h"
#include <fstream>
#include "camera_t.h"
#include "scene_t.h"
#include "render.h"

__global__ void init_kernel(vec3_t* d_framebuffer) {
    int thread_id = (int)(blockIdx.x * blockDim.x + threadIdx.x);
    if (thread_id >= NUM_PIXELS)
        return;

    d_framebuffer[thread_id] = vec3_t::make_zeros();
}

__global__ void render_kernel(camera_t* d_camera, scene_t* d_scene, vec3_t* d_framebuffer) {
    int thread_id = (int)(blockIdx.x * blockDim.x + threadIdx.x);
    if (thread_id >= NUM_PIXELS * SAMPLES_PER_PIXEL)
        return;

    int pixel_idx = thread_id / SAMPLES_PER_PIXEL;
    int x = pixel_idx % IMAGE_WIDTH;
    int y = pixel_idx / IMAGE_WIDTH;

    hiprandState rand_state;
    hiprand_init(0, thread_id, 0, &rand_state);
    float s = float(x) / float(IMAGE_WIDTH - 1);
    float t = 1.0f - float(y) / float(IMAGE_HEIGHT - 1);
    ray_t camera_ray = d_camera->get_ray(s, t);
    vec3_t color = get_color(*d_scene, camera_ray, rand_state);
    d_framebuffer[pixel_idx].atomic_add(color);
}

int main() {
    // camera
    vec3_t lookfrom(0.5f, 0.5f, 1.0f);
    vec3_t lookat(0.5f, 0.5f, 0.0f);
    vec3_t vup(0.0f, 1.0f, 0.0f);
    float vfov = 55.0f;
    float aspect_ratio = 1.0f;
    camera_t camera(lookfrom, lookat, vup, vfov, aspect_ratio);
    camera_t* d_camera;
    CHECK_CUDA(hipMalloc(&d_camera, sizeof(camera_t)));
    CHECK_CUDA(hipMemcpy(d_camera, &camera, sizeof(camera_t), hipMemcpyHostToDevice));

    // color
    vec3_t red(0.65f, 0.05f, 0.05f);
    vec3_t green(0.12f, 0.45f, 0.15f);
    vec3_t white(0.73f, 0.73f, 0.73f);
    vec3_t brown(0.62f, 0.57f, 0.54f);

    // scene
    std::vector<sphere_t> spheres;
    std::vector<trig_t> trigs;
    auto add_rectangle = [&](const vec3_t &p0, const vec3_t &p1, const vec3_t &p2,
                             const vec3_t &p3, const vec3_t &albedo) {
        trigs.emplace_back(p0, p1, p2, albedo);
        trigs.emplace_back(p2, p3, p0, albedo);
    };
    spheres.emplace_back(vec3_t(0.5f, 0.2f, -0.25f), 0.2f, brown);
    add_rectangle(vec3_t(0.0f, 0.0f, 0.0f),
                  vec3_t(0.0f, 1.0f, 0.0f),
                  vec3_t(0.0f, 1.0f, -1.0f),
                  vec3_t(0.0f, 0.0f, -1.0f),
                  red);
    add_rectangle(vec3_t(0.0f, 0.0f, -1.0f),
                  vec3_t(0.0f, 1.0f, -1.0f),
                  vec3_t(1.0f, 1.0f, -1.0f),
                  vec3_t(1.0f, 0.0f, -1.0f),
                  white);
    add_rectangle(vec3_t(1.0f, 0.0f, 0.0f),
                  vec3_t(1.0f, 1.0f, 0.0f),
                  vec3_t(1.0f, 1.0f, -1.0f),
                  vec3_t(1.0f, 0.0f, -1.0f),
                  green);
    add_rectangle(vec3_t(0.0f, 1.0f, 0.0f),
                  vec3_t(0.0f, 1.0f, -1.0f),
                  vec3_t(1.0f, 1.0f, -1.0f),
                  vec3_t(1.0f, 1.0f, 0.0f),
                  white);
    add_rectangle(vec3_t(0.0f, 0.0f, 0.0f),
                  vec3_t(0.0f, 0.0f, -1.0f),
                  vec3_t(1.0f, 0.0f, -1.0f),
                  vec3_t(1.0f, 0.0f, 0.0f),
                  white);
    scene_t scene{};
    scene.num_spheres = (int)spheres.size();
    CHECK_CUDA(hipMalloc(&scene.spheres, scene.num_spheres * sizeof(sphere_t)));
    CHECK_CUDA(hipMemcpy(scene.spheres, spheres.data(), scene.num_spheres * sizeof(sphere_t), hipMemcpyHostToDevice));
    scene.num_trigs = (int)trigs.size();
    CHECK_CUDA(hipMalloc(&scene.trigs, scene.num_trigs * sizeof(trig_t)));
    CHECK_CUDA(hipMemcpy(scene.trigs, trigs.data(), scene.num_trigs * sizeof(trig_t), hipMemcpyHostToDevice));
    scene.point_light = {vec3_t(0.95f, 0.95f, 0.3f), vec3_t(0.9f, 0.9f, 0.9f)};
    scene_t* d_scene;
    CHECK_CUDA(hipMalloc(&d_scene, sizeof(scene_t)));
    CHECK_CUDA(hipMemcpy(d_scene, &scene, sizeof(scene_t), hipMemcpyHostToDevice));

    // render
    vec3_t* d_framebuffer;
    CHECK_CUDA(hipMalloc(&d_framebuffer, NUM_PIXELS * sizeof(vec3_t)));
    init_kernel<<<(NUM_PIXELS + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(d_framebuffer);
    CHECK_CUDA(hipGetLastError());
    render_kernel<<<(NUM_PIXELS * SAMPLES_PER_PIXEL + BLOCK_SIZE - 1), BLOCK_SIZE>>>(d_camera, d_scene, d_framebuffer);
    CHECK_CUDA(hipGetLastError());

    // write framebuffer to file
    vec3_t framebuffer[NUM_PIXELS];
    CHECK_CUDA(hipMemcpy(framebuffer, d_framebuffer, NUM_PIXELS * sizeof(vec3_t), hipMemcpyDeviceToHost));
    std::ofstream image_fs("image.ppm");
    image_fs << "P3\n" << IMAGE_WIDTH << ' ' << IMAGE_HEIGHT << "\n255\n";
    for (int i = 0; i < IMAGE_HEIGHT; i++) {
        for (int j = 0; j < IMAGE_WIDTH; j++) {
            vec3_t color = framebuffer[i * IMAGE_WIDTH + j] / SAMPLES_PER_PIXEL;
            color.write_color(image_fs);
        }
    }

    return 0;
}
