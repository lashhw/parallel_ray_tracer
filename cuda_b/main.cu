#include "hip/hip_runtime.h"
#include <fstream>
#include "camera_t.h"
#include "scene_t.h"
#include "render.h"

__global__ void init_kernel(vec3_t* d_framebuffer, unsigned int image_width, unsigned int image_height) {
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= image_width || y >= image_height)
        return;

    d_framebuffer[y * image_width + x] = vec3_t::make_zeros();
}

__global__ void render_kernel(camera_t* d_camera, scene_t* d_scene, vec3_t* d_framebuffer,
                              unsigned int image_width, unsigned int image_height) {
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;
    unsigned int thread_id = z * gridDim.x * blockDim.x * gridDim.y * blockDim.y +
                             y * gridDim.x * blockDim.x + x;
    if (x >= image_width || y >= image_height || z >= SAMPLES_PER_PIXEL)
        return;

    hiprandState rand_state;
    hiprand_init(0, thread_id, 0, &rand_state);
    float s = float(x) / float(image_width - 1);
    float t = 1.0f - float(y) / float(image_height - 1);
    ray_t camera_ray = d_camera->get_ray(s, t);
    vec3_t color = get_color(*d_scene, camera_ray, rand_state);
    d_framebuffer[y * image_width + x].atomic_add(color);
}

int main() {
    // camera
    vec3_t lookfrom(0.5f, 0.5f, 1.0f);
    vec3_t lookat(0.5f, 0.5f, 0.0f);
    vec3_t vup(0.0f, 1.0f, 0.0f);
    float vfov = 55.0f;
    float aspect_ratio = 1.0f;
    camera_t camera(lookfrom, lookat, vup, vfov, aspect_ratio);
    camera_t* d_camera;
    CHECK_CUDA(hipMalloc(&d_camera, sizeof(camera_t)));
    CHECK_CUDA(hipMemcpy(d_camera, &camera, sizeof(camera_t), hipMemcpyHostToDevice));

    // color
    vec3_t red(0.65f, 0.05f, 0.05f);
    vec3_t green(0.12f, 0.45f, 0.15f);
    vec3_t white(0.73f, 0.73f, 0.73f);
    vec3_t brown(0.62f, 0.57f, 0.54f);

    // scene
    std::vector<sphere_t> spheres;
    std::vector<trig_t> trigs;
    auto add_rectangle = [&](const vec3_t &p0, const vec3_t &p1, const vec3_t &p2,
                             const vec3_t &p3, const vec3_t &albedo) {
        trigs.emplace_back(p0, p1, p2, albedo);
        trigs.emplace_back(p2, p3, p0, albedo);
    };
    spheres.emplace_back(vec3_t(0.5f, 0.2f, -0.25f), 0.2f, brown);
    add_rectangle(vec3_t(0.0f, 0.0f, 0.0f),
                  vec3_t(0.0f, 1.0f, 0.0f),
                  vec3_t(0.0f, 1.0f, -1.0f),
                  vec3_t(0.0f, 0.0f, -1.0f),
                  red);
    add_rectangle(vec3_t(0.0f, 0.0f, -1.0f),
                  vec3_t(0.0f, 1.0f, -1.0f),
                  vec3_t(1.0f, 1.0f, -1.0f),
                  vec3_t(1.0f, 0.0f, -1.0f),
                  white);
    add_rectangle(vec3_t(1.0f, 0.0f, 0.0f),
                  vec3_t(1.0f, 1.0f, 0.0f),
                  vec3_t(1.0f, 1.0f, -1.0f),
                  vec3_t(1.0f, 0.0f, -1.0f),
                  green);
    add_rectangle(vec3_t(0.0f, 1.0f, 0.0f),
                  vec3_t(0.0f, 1.0f, -1.0f),
                  vec3_t(1.0f, 1.0f, -1.0f),
                  vec3_t(1.0f, 1.0f, 0.0f),
                  white);
    add_rectangle(vec3_t(0.0f, 0.0f, 0.0f),
                  vec3_t(0.0f, 0.0f, -1.0f),
                  vec3_t(1.0f, 0.0f, -1.0f),
                  vec3_t(1.0f, 0.0f, 0.0f),
                  white);
    scene_t scene{};
    scene.num_spheres = (int)spheres.size();
    CHECK_CUDA(hipMalloc(&scene.spheres, scene.num_spheres * sizeof(sphere_t)));
    CHECK_CUDA(hipMemcpy(scene.spheres, spheres.data(), scene.num_spheres * sizeof(sphere_t), hipMemcpyHostToDevice));
    scene.num_trigs = (int)trigs.size();
    CHECK_CUDA(hipMalloc(&scene.trigs, scene.num_trigs * sizeof(trig_t)));
    CHECK_CUDA(hipMemcpy(scene.trigs, trigs.data(), scene.num_trigs * sizeof(trig_t), hipMemcpyHostToDevice));
    scene.point_light = {vec3_t(0.95f, 0.95f, 0.3f), vec3_t(0.9f, 0.9f, 0.9f)};
    scene_t* d_scene;
    CHECK_CUDA(hipMalloc(&d_scene, sizeof(scene_t)));
    CHECK_CUDA(hipMemcpy(d_scene, &scene, sizeof(scene_t), hipMemcpyHostToDevice));

    // render
    unsigned int image_width = 600;
    unsigned int image_height = 600;

    vec3_t* d_framebuffer;
    CHECK_CUDA(hipMalloc(&d_framebuffer, image_height * image_width * sizeof(vec3_t)));
    {
        dim3 block_size(BLOCK_SIZE_X, BLOCK_SIZE_Y);
        dim3 grid_size((image_width + block_size.x - 1) / block_size.x,
                       (image_height + block_size.y - 1) / block_size.y);
        init_kernel<<<grid_size, block_size>>>(d_framebuffer, image_width, image_height);
        CHECK_CUDA(hipGetLastError());
    }
    {
        dim3 block_size(BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z);
        dim3 grid_size((image_width + block_size.x - 1) / block_size.x,
                       (image_height + block_size.y - 1) / block_size.y,
                       (SAMPLES_PER_PIXEL + block_size.z - 1) / block_size.z);
        render_kernel<<<grid_size, block_size>>>(d_camera, d_scene, d_framebuffer, image_width, image_height);
        CHECK_CUDA(hipGetLastError());
    }

    // write framebuffer to file
    vec3_t framebuffer[image_height * image_width];
    CHECK_CUDA(hipMemcpy(framebuffer, d_framebuffer, image_height * image_width * sizeof(vec3_t), hipMemcpyDeviceToHost));
    std::ofstream image_fs("image.ppm");
    image_fs << "P3\n" << image_width << ' ' << image_height << "\n255\n";
    for (int i = 0; i < image_height; i++) {
        for (int j = 0; j < image_width; j++) {
            vec3_t color = framebuffer[i * image_width + j] / SAMPLES_PER_PIXEL;
            color.write_color(image_fs);
        }
    }

    return 0;
}
